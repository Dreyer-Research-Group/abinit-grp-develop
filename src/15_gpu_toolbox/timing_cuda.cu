/* prt_utils_rec.cu */

/*
 * Copyright (C) 2008-2018 ABINIT Group (MMancini)
 *
 * This file is part of the ABINIT software package. For license information,
 * please see the COPYING file in the top-level directory of the ABINIT source
 * distribution.
 *
 */

#include "cuda_common.h"
#include "cuda_header.h"
#include "cuda_rec_head.h"

/*=========================================================================*/
/*_________________________TIMING IN CUDA ROUTINES_________________________*/
/*=========================================================================*/
/* This file contains some basic utils from the time measuration in
 * cuda subroutines. A more particular version is contained in 
 * prt_utils_rec.cu (to put together)
*/


/*~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~*/
/*~~~~~~~~~~~~~~~~~~~~~~~~ INTERFACE WITH FORTRAN ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~*/
/*~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~*/
extern "C" __host__
void start_cuda_tm_(hipEvent_t* start,hipEvent_t* stop)
{
   hipEventCreate(start);
   hipEventCreate(stop);
   hipEventRecord(*start,0);
   return;
};

extern "C" __host__
void stop_cuda_tm_(hipEvent_t* stop)
{
   hipEventRecord(*stop,0);
   hipEventSynchronize(*stop);
   printf("stop %d\n",*stop);
   return;
}

extern "C" __host__
void calc_cuda_time_(hipEvent_t* stop,hipEvent_t* start,float* time_ms)
{
#if defined HAVE_GPU_CUDA3
   hipDeviceSynchronize();
#else
   hipDeviceSynchronize();
#endif   
   *time_ms = 0.;
   stop_cuda_tm_(stop);
   hipEventElapsedTime(time_ms,*start,*stop);
   printf("stop %d\n",*start);
   printf("stop %d\n",*stop);
   printf("stop %f\n",time_ms);
   hipEventDestroy(*start);
   hipEventDestroy(*stop);
   return ;
}
